#include "hip/hip_runtime.h"
#include"cudaImageAdd.h"
#include <hip/hip_runtime.h>
#include ""
#include "hip/hip_fp16.h"
#include <iostream>
using namespace std;

//bool cudaImageAdd::init()
//{
//	��1�� Build input image host memory and device memory
//	this->memorySize = imgHeight * imgWidth * batchSize;
//	this->renderImg_host  = new float[memorySize];
//	hipMalloc((void**)&this->renderImg_device, memorySize*sizeof(float));
//
//	****Initialize the host memory
//	//memset(this->renderImg_host, 0, memorySize*sizeof(float));
//	for (int i = 0; i < imgHeight*imgWidth*batchSize;i++)
//	{
//		renderImg_host[i] = 1;
//	}
//
//	****Copy the input image host value to the device
//	hipError_t state = hipMemcpy(this->renderImg_device, this->renderImg_host, this->memorySize * sizeof(float), hipMemcpyHostToDevice);
//	cout << "Input image error��" << state << endl;
//
//	//float* test = new float[2048 * 2048 * 13];
//	//hipMemcpy(test, renderImg_device, memorySize * sizeof(float), hipMemcpyDeviceToHost);
//
//	****Build output image host memory, device memory
//	this->renderImgOut_host = new float[imgHeight*imgWidth];
//	hipMalloc((void**)&this->renderImgOut_device, imgHeight*imgWidth * sizeof(float));
//
//	return true;
//}
//
//bool cudaImageAdd::deinit()
//{
//	****Free host memory
//	delete[] this->renderImg_host;
//	hipError_t state = hipFree(this->renderImg_device);
//	cout << "Input image device memory release error��" << state << endl;
//
//	delete[] this->renderImgOut_host;
//	state =  hipFree(this->renderImgOut_device);
//	cout << "Output image device memory release error��" << state << endl;
//	return true; 
//}
//
//bool cudaImageAdd::render() 
//{
//	��2�� Addition and subtraction of data
//	int threadPerBlock = 256;
//	int blockNums = (imgHeight * imgWidth + threadPerBlock  -1)/ threadPerBlock;
//
//	for(int i = 0;i<10;i++)
//		imgAdd <<<blockNums, threadPerBlock >>> (renderImg_device, renderImgOut_device, imgWidth, imgHeight, batchSize);
//
//	��3�� Copy the calculated data to the host
//	hipError_t state = hipMemcpy(this->renderImgOut_host, this->renderImgOut_device, imgHeight * imgWidth * sizeof(float), hipMemcpyDeviceToHost);
//
//	cout << "Output image error��" << state << endl;
//	printf("pause");
//	return true;
//}

__global__ void imgAdd(float* renderImg_device, float* renderImgOut_device, int imgWidth, int imgHeight, int batchSize, bool fp16)
{
	int blockId = blockIdx.x;
	int id = blockId * blockDim.x + threadIdx.x;

	if (id < imgWidth * imgHeight)
	{
		for (int i = 0; i < batchSize; i++)
		{
			if (!fp16)
				renderImgOut_device[id] += renderImg_device[id + i * imgWidth*imgHeight];
			else
				renderImgOut_device[id] += __half2float(renderImgOut_device[id + i * imgWidth*imgHeight]);		// fp16
		}
	}
}

void imgAddTop(float* renderImg_device, float* renderImgOut_device, int imgWidth, int imgHeight, int batchSize,bool fp16)
{
	int threadPerBlock = 256;
	int blockNums = (imgHeight * imgWidth + threadPerBlock - 1) / threadPerBlock;
	imgAdd << <blockNums, threadPerBlock >> > (renderImg_device, renderImgOut_device, imgWidth, imgHeight, batchSize,fp16);
}



//int main_cudaImageAdd00()
//{
//	cudaImageAdd add;
//	add.init();
//	add.render();
//	add.deinit();
//	return 0;
//}