#include "hip/hip_runtime.h"
#include"cudaImageAdd.h"
#include <hip/hip_runtime.h>
#include ""
#include "hip/hip_fp16.h"
#include <iostream>
using namespace std;

//bool cudaImageAdd::init()
//{
//	// ��������ͼ�������ڴ桢�豸�ڴ�
//	this->memorySize = imgHeight * imgWidth * batchSize;
//	this->renderImg_host  = new float[memorySize];
//	hipMalloc((void**)&this->renderImg_device, memorySize*sizeof(float));
//
//	// ��ʼ�������ڴ�
//	//memset(this->renderImg_host, 0, memorySize*sizeof(float));
//	for (int i = 0; i < imgHeight*imgWidth*batchSize;i++)
//	{
//		renderImg_host[i] = 1;
//	}
//
//	// ������ͼ��������ֵ�������豸
//	hipError_t state = hipMemcpy(this->renderImg_device, this->renderImg_host, this->memorySize * sizeof(float), hipMemcpyHostToDevice);
//	cout << "����ͼ���Ƿ����" << state << endl;
//
//	//float* test = new float[2048 * 2048 * 13];
//	//hipMemcpy(test, renderImg_device, memorySize * sizeof(float), hipMemcpyDeviceToHost);
//
//	// �������ͼ�������ڴ棬�豸�ڴ�
//	this->renderImgOut_host = new float[imgHeight*imgWidth];
//	hipMalloc((void**)&this->renderImgOut_device, imgHeight*imgWidth * sizeof(float));
//
//
//	return true;
//}
//
//bool cudaImageAdd::deinit()
//{
//	// �ͷ������ڴ�
//	delete[] this->renderImg_host;
//	hipError_t state = hipFree(this->renderImg_device);
//	cout << "����ͼ���豸�ڴ��ͷ��Ƿ����" << state << endl;
//
//	delete[] this->renderImgOut_host;
//	state =  hipFree(this->renderImgOut_device);
//	cout << "���ͼ���豸�ڴ��ͷ��Ƿ����" << state << endl;
//	return true; 
//}
//
//bool cudaImageAdd::render() 
//{
//	//��2�� ���ݵļӼ�
//	int threadPerBlock = 256;
//	int blockNums = (imgHeight * imgWidth + threadPerBlock  -1)/ threadPerBlock;
//
//	for(int i = 0;i<10;i++)
//		imgAdd <<<blockNums, threadPerBlock >>> (renderImg_device, renderImgOut_device, imgWidth, imgHeight, batchSize);
//
//	//��3�� ���������ݿ���������
//	hipError_t state = hipMemcpy(this->renderImgOut_host, this->renderImgOut_device, imgHeight * imgWidth * sizeof(float), hipMemcpyDeviceToHost);
//
//	cout << "���ͼ���Ƿ����" << state << endl;
//	printf("pause");
//	return true;
//}

__global__ void imgAdd(float* renderImg_device, float* renderImgOut_device, int imgWidth, int imgHeight, int batchSize, bool fp16)
{
	int blockId = blockIdx.x;
	int id = blockId * blockDim.x + threadIdx.x;

	if (id < imgWidth * imgHeight)
	{
		for (int i = 0; i < batchSize; i++)
		{
			if (!fp16)
				renderImgOut_device[id] += renderImg_device[id + i * imgWidth*imgHeight];
			else
				renderImgOut_device[id] += __half2float(renderImgOut_device[id + i * imgWidth*imgHeight]);		// fp16
		}
	}
}

void imgAddTop(float* renderImg_device, float* renderImgOut_device, int imgWidth, int imgHeight, int batchSize,bool fp16)
{
	int threadPerBlock = 256;
	int blockNums = (imgHeight * imgWidth + threadPerBlock - 1) / threadPerBlock;
	imgAdd << <blockNums, threadPerBlock >> > (renderImg_device, renderImgOut_device, imgWidth, imgHeight, batchSize,fp16);
}



//int main_cudaImageAdd00()
//{
//	cudaImageAdd add;
//	add.init();
//	add.render();
//	add.deinit();
//	return 0;
//}